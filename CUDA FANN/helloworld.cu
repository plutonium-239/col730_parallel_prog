
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void cuda_print(){
	printf("Hello from GPU from thread [%d,%d]\n",blockIdx.x, threadIdx.x);
}

int main (){
	int driverVersion;
	int runtimeVersion;

	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("%d, %d\n",driverVersion, runtimeVersion);

	cuda_print<<<10,1>>>();
	//cudaDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		printf("Error %s\n",hipGetErrorString(err));

	return 0;
}
