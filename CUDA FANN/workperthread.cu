#include "hip/hip_runtime.h"
/*
Fast Artificial Neural Network Library (fann)
Copyright (C) 2003-2016 Steffen Nissen (steffen.fann@gmail.com)

This library is free software; you can redistribute it and/or
modify it under the terms of the GNU Lesser General Public
License as published by the Free Software Foundation; either
version 2.1 of the License, or (at your option) any later version.

This library is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
Lesser General Public License for more details.

You should have received a copy of the GNU Lesser General Public
License along with this library; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
*/

#include <stdio.h>
#include "fann.h"
#include <cstdlib>

__global__ void pass_layer_i(struct fann *ann, struct fann_neuron *neurons, 
	fann_type *dev_weights, struct fann_neuron *dev_last_layer_neurons, int total_work);

__global__ void layer_activations(struct fann_neuron *dev_neurons, int total_work);

__global__ void calc_mse(struct fann *ann, struct fann_neuron *dev_last_layer_begin, fann_type *dev_output, 
	fann_type *error_begin, int last_layer_first_neuron_number, int total_work);

__device__ fann_type sigmoid_gradient(fann_type x, fann_type steepness);

__global__ void fann_backprop_layer(fann_type *dev_weights, struct fann_neuron *dev_neurons, int first_neuron_number,
	int last_layer_first_neuron_number, fann_type *error_begin, int total_work);

__global__ void fann_backprop_layer_complete(fann_type *error_begin, struct fann_neuron *dev_last_layer_neurons, 
	int last_layer_first_neuron_number, int total_work);

__global__ void update_slopes(struct fann_neuron *dev_neurons, struct fann_neuron *dev_last_layer_neurons, int first_neuron_number, 
	fann_type *dev_slopes, fann_type *dev_errors, int total_work);

__global__ void pr(fann_type *error_begin, int last_layer_first_neuron_number, int num_connections);

__global__ void vector_multiply(fann_type *neuron_sum_ptr, fann_type *dev_weights, struct fann_neuron *dev_neurons);

__global__ void gather(fann_type *arr, int N, fann_type *sum);

FANN_EXTERNAL fann_type *FANN_API fann_run(struct fann *ann, fann_type *input, struct fann *dev_ann, fann_type *dev_weights);

float fann_train_epoch_irpropm_custom(struct fann *ann, struct fann_train_data *data);

void fann_backpropagate_MSE_custom(struct fann *ann, struct fann *dev_ann, fann_type *dev_weights, fann_type *dev_errors);

void fann_compute_MSE_custom(struct fann *ann, fann_type *desired_output, fann_type *desired_output_host, fann_type *dev_errors, struct fann *dev_ann);

void fann_update_slopes_batch_custom(struct fann *ann, struct fann_layer *layer_begin,
                              struct fann_layer *layer_end, fann_type *dev_slopes, fann_type *dev_errors);

void fann_update_weights_irpropm(struct fann *ann, unsigned int first_weight,
                                 unsigned int past_end);

fann_type fann_update_MSE(struct fann *ann, struct fann_neuron *neuron, fann_type neuron_diff);


#define check(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int scale_threads = 1; // calling all kernels with num_threads/scale_threads to see how scaling occurs

int main(int argc, char* argv[])
{
  if (argc < 2)
    printf("Needs at least dataset choice as argument!\n");
  
  int choice = atoi(argv[1]);
	unsigned int num_layers = 3;
	unsigned int num_neurons_hidden = 32;
	float desired_error = (const float) 0.0001;
  unsigned int max_epochs = 300;
	unsigned int epochs_between_reports = 1;
	struct fann *ann;
	struct fann_train_data *train_data, *test_data;

  if (choice == 0) {
    printf("Using mushroom dataset\n");
    num_layers = 3;
    num_neurons_hidden = 32;
    desired_error = 0.0001;
    max_epochs = 30;
    train_data = fann_read_train_from_file("../datasets/mushroom.train");
    test_data = fann_read_train_from_file("../datasets/mushroom.test");
    ann = fann_create_standard(num_layers,
              train_data->num_input, num_neurons_hidden, train_data->num_output);
  }
  else if (choice == 1) {
    printf("Using gene dataset\n");
    num_layers = 3;
    num_neurons_hidden = 512;
    desired_error = 0.0001;
    train_data = fann_read_train_from_file("../datasets/gene.train");
    test_data = fann_read_train_from_file("../datasets/gene.test");
    ann = fann_create_standard(num_layers,
              train_data->num_input, num_neurons_hidden, train_data->num_output);
  }
  else if (choice == 2) {
    printf("Using soybean dataset\n");
    num_layers = 3;
    num_neurons_hidden = 64;
    desired_error = 0.001;
    train_data = fann_read_train_from_file("../datasets/soybean.train");
    test_data = fann_read_train_from_file("../datasets/soybean.test");
    ann = fann_create_standard(num_layers,
              train_data->num_input, num_neurons_hidden, train_data->num_output);
  }
  else if (choice == 3) {
    printf("Using pumadyn dataset\n");
    num_layers = 3;
    num_neurons_hidden = 32;
    max_epochs = 50;
    desired_error = 0.01;
    train_data = fann_read_train_from_file("../datasets/pumadyn-32fm.train");
    test_data = fann_read_train_from_file("../datasets/pumadyn-32fm.test");
  	ann = fann_create_standard(num_layers,
  					  train_data->num_input, num_neurons_hidden, train_data->num_output);
  }
  
  if (argc == 3) {
    scale_threads = atoi(argv[2]);
    printf("Using scale_threads = %d\n", scale_threads);
  }

  unsigned int i = 0;

  
	printf("Creating network with layers (%u,%u,%u)\n", train_data->num_input, num_neurons_hidden, train_data->num_output);

	
  ann->train_errors = (fann_type *)calloc(ann->total_neurons, sizeof(fann_type));
  if (ann->train_errors == NULL) {
    fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
    return 1;
  }

  ann->train_slopes = (fann_type *)calloc(ann->total_connections_allocated, sizeof(fann_type));
  if (ann->train_slopes == NULL) {
    fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
    return -1;
  }
  
	printf("Training network.\n");

	fann_set_activation_function_hidden(ann, FANN_SIGMOID_SYMMETRIC);
	fann_set_activation_function_output(ann, FANN_SIGMOID);

	/*fann_set_training_algorithm(ann, FANN_TRAIN_INCREMENTAL); */

	fann_train_on_data(ann, train_data, max_epochs, epochs_between_reports, desired_error);

	printf("Testing network.\n");

	fann_reset_MSE(ann);
	for(i = 0; i < fann_length_train_data(test_data); i++)
	{
		fann_test(ann, test_data->input[i], test_data->output[i]);
	}
	
	printf("MSE error on test data: %f\n", fann_get_MSE(ann));

	printf("Saving network.\n");

	fann_save(ann, "mushroom_float.net");

	printf("Cleaning up.\n");
	fann_destroy_train(train_data);
	fann_destroy_train(test_data);
	fann_destroy(ann);

	return 0;
}

FANN_EXTERNAL void FANN_API fann_train_on_data(struct fann *ann, struct fann_train_data *data,
                                               unsigned int max_epochs,
                                               unsigned int epochs_between_reports,
                                               float desired_error) {
  float error;
  unsigned int i;
  int desired_error_reached;

#ifdef DEBUG
  printf("Training with %s\n", FANN_TRAIN_NAMES[ann->training_algorithm]);
#endif

  if (epochs_between_reports && ann->callback == NULL) {
    printf("Max epochs %8d. Desired error: %.10f.\n", max_epochs, desired_error);
  }
  printf("USING CUSTOM DEFINITION\n");

  for (i = 1; i <= max_epochs; i++) {
    /*
     * train
     */
    error = fann_train_epoch_irpropm_custom(ann, data);
    desired_error_reached = fann_desired_error_reached(ann, desired_error);

    /*
     * print current output
     */
    if (epochs_between_reports && (i % epochs_between_reports == 0 || i == max_epochs || i == 1 ||
                                   desired_error_reached == 0)) {
      if (ann->callback == NULL) {
        printf("Epochs     %8d. Current error: %.10f. Bit fail %d.\n", i, error, ann->num_bit_fail);
      } else if (((*ann->callback)(ann, data, max_epochs, epochs_between_reports, desired_error,
                                   i)) == -1) {
        /*
         * you can break the training by returning -1
         */
        break;
      }
    }

    if (desired_error_reached == 0) break;
  }
}

float fann_train_epoch_irpropm_custom(struct fann *ann, struct fann_train_data *data) {
  unsigned int i;
  if (ann->prev_train_slopes == NULL) {
  	fann_clear_train_arrays(ann);
  }

  fann_reset_MSE(ann);

  struct fann *dev_ann;
	fann_type *dev_weights, *dev_errors, *dev_output, *dev_slopes;
	check(hipMalloc((void **)&dev_ann, sizeof(struct fann)));
	check(hipMalloc((void **)&dev_weights, ann->total_connections*sizeof(fann_type)));
	check(hipMalloc((void **)&dev_errors, ann->total_neurons*sizeof(fann_type)));
	check(hipMalloc((void **)&dev_output, data->num_data*data->num_output*sizeof(fann_type)));
	check(hipMalloc((void **)&dev_slopes, ann->total_connections_allocated*sizeof(fann_type)));



	// printf("%d vs %d*%d\n", sizeof(ann->weights), ann->total_connections, sizeof(fann_type));
	check(hipMemcpy(dev_ann, ann, sizeof(struct fann), hipMemcpyHostToDevice));
	check(hipMemcpy(dev_weights, ann->weights, ann->total_connections*sizeof(fann_type), hipMemcpyHostToDevice));
	// check(hipMemcpy(dev_weights, ann->weights, ann->total_connections*sizeof(fann_type), hipMemcpyHostToDevice));





  for (i = 0; i < data->num_data; i++) {
  	fann_run(ann, data->input[i], dev_ann, dev_weights);
  	// fann_type *out = data->output[i];
  	// printf("\n%f %f | ", *out, *(out+1));
		check(hipMemcpy(&dev_output[i], data->output[i], data->num_output*sizeof(fann_type), hipMemcpyHostToDevice));
    fann_compute_MSE_custom(ann, &dev_output[i], data->output[i], dev_errors, dev_ann);
    fann_backpropagate_MSE_custom(ann, dev_ann, dev_weights, dev_errors);
    fann_update_slopes_batch_custom(ann, ann->first_layer + 1, ann->last_layer - 1, dev_slopes, dev_errors);
  }

	check(hipMemcpy(ann->weights, dev_weights, ann->total_connections*sizeof(fann_type), hipMemcpyDeviceToHost));
	check(hipMemcpy(ann->train_slopes, dev_slopes, ann->total_connections_allocated*sizeof(fann_type), hipMemcpyDeviceToHost));

  // printf("%p\n", ann->prev_steps);
  // for (i = 0; i < ann->total_connections; ++i)
  // {
	 //  printf("%d", *(ann->prev_steps+i));
  // }
  fann_update_weights_irpropm(ann, 0, ann->total_connections);

  hipFree(dev_ann);
  hipFree(dev_weights);
  hipFree(dev_output);
  hipFree(dev_errors);
  hipFree(dev_slopes);
  
  return fann_get_MSE(ann);
}

FANN_EXTERNAL fann_type *FANN_API fann_run(struct fann *ann, fann_type *input, struct fann *dev_ann, fann_type *dev_weights) {
  struct fann_neuron *neuron_it, *last_neuron, *neurons, **neuron_pointers;
  unsigned int i, num_input, num_output;
  fann_type *output;
  struct fann_layer *layer_it, *last_layer;

  /* store some variabels local for fast access */
  struct fann_neuron *first_neuron = ann->first_layer->first_neuron;

#ifdef FIXEDFANN
  int multiplier = ann->multiplier;
  unsigned int decimal_point = ann->decimal_point;

  /* values used for the stepwise linear sigmoid function */
  fann_type r1 = 0, r2 = 0, r3 = 0, r4 = 0, r5 = 0, r6 = 0;
  fann_type v1 = 0, v2 = 0, v3 = 0, v4 = 0, v5 = 0, v6 = 0;

  fann_type last_steepness = 0;
  unsigned int last_activation_function = 0;
#else
  fann_type max_sum = 0;
#endif

  /* first set the input */
  num_input = ann->num_input;
  for (i = 0; i != num_input; i++) {
#ifdef FIXEDFANN
    if (fann_abs(input[i]) > multiplier) {
      printf(
          "Warning input number %d is out of range -%d - %d with value %d, integer overflow may "
          "occur.\n",
          i, multiplier, multiplier, input[i]);
    }
#endif
    first_neuron[i].value = input[i];
  }
  /* Set the bias neuron in the input layer */
#ifdef FIXEDFANN
  (ann->first_layer->last_neuron - 1)->value = multiplier;
#else
  (ann->first_layer->last_neuron - 1)->value = 1;
#endif

  last_layer = ann->last_layer;

  for (layer_it = ann->first_layer + 1; layer_it != last_layer; layer_it++) {
  	int neurons_in_layer = layer_it->last_neuron - layer_it->first_neuron -1;
  	int neurons_in_prev_layer = (layer_it-1)->last_neuron - (layer_it-1)->first_neuron -1;

  	// printf("Neurons in layer_it:%d, prev layer:%d\n", neurons_in_layer, neurons_in_prev_layer);
  // 	hipError_t err1 = hipGetLastError();
		// if(err1 != hipSuccess)
		// 	printf("Error %s\n",hipGetErrorString(err1));
		struct fann_neuron *dev_neurons, *dev_last_layer_neurons;
		check(hipMalloc((void **)&dev_last_layer_neurons, neurons_in_prev_layer*sizeof(struct fann_neuron)));
  	check(hipMemcpy(dev_last_layer_neurons, (layer_it-1)->first_neuron, neurons_in_prev_layer*sizeof(struct fann_neuron), hipMemcpyHostToDevice));

  	check(hipMalloc((void **)&dev_neurons, neurons_in_layer*sizeof(struct fann_neuron)));
  	check(hipMemcpy(dev_neurons, layer_it->first_neuron, neurons_in_layer*sizeof(struct fann_neuron), hipMemcpyHostToDevice));
		// printf("dev_n: %p to %p\n",dev_neurons, dev_neurons+neurons_in_layer);
		// hipError_t err2 = hipGetLastError();
		// if(err2 != hipSuccess)
		// 	printf("Error 2 %s\n",hipGetErrorString(err2));

  	// for (neuron_it = layer_it->first_neuron, i=0; neuron_it != last_neuron; neuron_it++) {
  	// 	hipMemcpy(dev_neurons + i, neuron_it, sizeof(struct fann_neuron), hipMemcpyHostToDevice);
  	// 	hipError_t err1 = hipGetLastError();
			// if(err1 != hipSuccess)
			// 	printf("Error 2 %s\n",hipGetErrorString(err1));
  	// 	i++;
  	// }
  	// printf("layer %d\n", neurons_in_layer);
    // printf("\n");
    // printf("\n");
    pass_layer_i<<<1,(int) (neurons_in_layer/scale_threads)>>>(dev_ann, dev_neurons, dev_weights, 
      dev_last_layer_neurons, neurons_in_layer);
    // pass_layer_i<<<1,1>>>(dev_ann, dev_neurons, dev_weights, dev_last_layer_neurons, neurons_in_layer);
    // hipDeviceSynchronize();
    // printf("\nvs\n");
    // layer_activations<<<1,1>>>(dev_neurons, neurons_in_layer);
    // layer_activations<<<1, (int) (neurons_in_layer/scale_threads)>>>(dev_neurons, neurons_in_layer);
    // printf("\n");
  	check(hipMemcpy(layer_it->first_neuron, dev_neurons, neurons_in_layer*sizeof(struct fann_neuron), hipMemcpyDeviceToHost));
		hipError_t err4 = hipGetLastError();
		if(err4 != hipSuccess)
			printf("Error 4 %s\n",hipGetErrorString(err4));
  	
   	//  for (neuron_it = layer_it->first_neuron, i=0; neuron_it != last_neuron; neuron_it++) {
  	// 	hipMemcpy(neuron_it, dev_neurons + i, sizeof(struct fann_neuron), hipMemcpyDeviceToHost);
  	// 	i++;
  	// }
  	hipFree(dev_neurons);
  	hipFree(dev_last_layer_neurons);
    }


  /* set the output */
  output = ann->output;
  num_output = ann->num_output;
  neurons = (ann->last_layer - 1)->first_neuron;
  for (i = 0; i != num_output; i++) {
    output[i] = neurons[i].value;
  }
  return ann->output;
}

__global__ void vector_multiply(fann_type *neuron_sum_ptr, fann_type *dev_weights, struct fann_neuron *dev_last_layer_neurons) {
  neuron_sum_ptr[threadIdx.x] = fann_mult(dev_weights[threadIdx.x], dev_last_layer_neurons[threadIdx.x].value);
  // printf("at %p + %d : %.3f * %.3f = %.3f \n", neuron_sum_ptr, threadIdx.x, dev_weights[threadIdx.x], 
    // dev_last_layer_neurons[threadIdx.x].value, neuron_sum_ptr[threadIdx.x]);
  // printf("-");
}

__global__ void gather(fann_type *temp_outs, int N, fann_type *sum) {
  // fann_type temp_sum = 0;
  for (int i = 0; i < N-1; ++i)
  {
    // printf("%p (%p + %d) received = %.3f\n", temp_outs+i, temp_outs, i, temp_outs[i]);
    *sum += temp_outs[i];
  }
}

__global__ void pass_layer_i(struct fann *ann, struct fann_neuron *dev_neurons, fann_type *dev_weights,
		struct fann_neuron *dev_last_layer_neurons, int total_work) {
  int work_per_thread = total_work/blockDim.x;
  for (int work = 0; work < work_per_thread; ++work)
  {
    unsigned int activation_function;
    unsigned int i, num_connections;
  	struct fann_neuron *neuron_it, *last_neuron, *neurons, **neuron_pointers;
    fann_type steepness;
  	fann_type *weights, neuron_sum, max_sum;
    

    neuron_it = dev_neurons + threadIdx.x + work*work_per_thread; // COALESCED (also change call to vector_multiply)
    // neuron_it = dev_neurons + threadIdx.x*work_per_thread + work; // NOT COALESCED

    // printf("Kernel %p is running, %p\n", neuron_it, layer_it->first_neuron);

    neuron_it->value = 1;

    neuron_sum = 0;
    num_connections = neuron_it->last_con - neuron_it->first_con;
    weights = dev_weights + neuron_it->first_con;
    // printf("ann->wts:%p vs dev_wts:%p vs neuron_it->first_con:%p\n",ann->weights, dev_weights, neuron_it->first_con);

    // neurons = (layer_it - 1)->first_neuron;
    neurons = dev_last_layer_neurons;
    // printf("wts:%p, neur:%p, dev_neur:%p\n", weights, neurons, dev_neurons);


    // if (!use_vector) {
      for(i = 0;i != num_connections-1; i++){
        neuron_sum += fann_mult(weights[i], neurons[i].value);
      }
    // }
    // else {
    //   fann_type *temp_outs;
    //   hipMalloc((void **)&temp_outs, (num_connections-1)*sizeof(fann_type));
    //   fann_type *temp_sum;
    //   hipMalloc((void **)&temp_sum, sizeof(fann_type));

    //   vector_multiply<<<1, num_connections-1>>>(temp_outs, dev_weights, dev_last_layer_neurons);
    //   gather<<<1,1>>>(temp_outs, num_connections-1, temp_sum);
    //   hipFree(temp_outs);
    //   neuron_sum = *temp_sum;
    //   hipFree(temp_sum);
    // }

    

    // printf("%d %.3f vs %.3f | ", threadIdx.x + work*work_per_thread, neuron_sum, temp_sum);
    /* unrolled loop start */
    // i = num_connections & 3; /* same as modulo 4 */
    // // printf("starting %p at %d\n", neuron_it, i);
    // switch (i) {
    //   case 3:
    //     neuron_sum += fann_mult(weights[2], neurons[2].value);
    //   case 2:
    //     neuron_sum += fann_mult(weights[1], neurons[1].value);
    //   case 1:
    //     neuron_sum += fann_mult(weights[0], neurons[0].value);
    //   case 0:
    //     break;
    // }

    // for (; i < num_connections-4; i +=4) {
    // 	// if (i<0) {
    //   // 	printf("i %u: ", i+3);
    //   // 	printf("w %f, ", weights[i+3]);
    //   // 	printf("n %f", neurons[i+3].value);
    // 	// }
    //   neuron_sum += fann_mult(weights[i], neurons[i].value);
    //   neuron_sum += fann_mult(weights[i + 1], neurons[i + 1].value);
    //   neuron_sum += fann_mult(weights[i + 2], neurons[i + 2].value);
    //   neuron_sum += fann_mult(weights[i + 3], neurons[i + 3].value);
    // }
    // for (i=0; i != num_connections-1; i ++) {
    //   neuron_sum += fann_mult(weights[i], neurons[i].value);
    // }

    /* unrolled loop end */

    
     
    // neuron_it->sum = neuron_sum;
    
  // #else
    activation_function = neuron_it->activation_function;
    steepness = neuron_it->activation_steepness;
    neuron_sum = fann_mult(steepness, neuron_sum);

    max_sum = 150 / steepness;
    if (neuron_sum > max_sum)
      neuron_sum = max_sum;
    else if (neuron_sum < -max_sum)
      neuron_sum = -max_sum;

    neuron_it->sum = neuron_sum;

    fann_activation_switch(activation_function, neuron_sum, neuron_it->value);
  // #endif
    // printf("ended %p\n", neuron_it);
    // }
    // hipMemcpy(neuron_it, dev_neuron_it, sizeof(fann_neuron), hipMemcpyDeviceToHost);
    // hipFree(dev_neuron_it);
  }
}

__global__ void layer_activations(struct fann_neuron *dev_neurons, int total_work) {
  int work_per_thread = total_work/blockDim.x;
  for (int work = 0; work < work_per_thread; ++work)
  {
    struct fann_neuron *neuron_it;
    fann_type steepness, max_sum;
    unsigned int activation_function;

    neuron_it = dev_neurons + threadIdx.x + work*work_per_thread; // COALESCED (also change call to vector_multiply)

    activation_function = neuron_it->activation_function;
    steepness = neuron_it->activation_steepness;

    neuron_it->sum = fann_mult(steepness, neuron_it->sum);
    max_sum = 150 / steepness;
    if (neuron_it->sum > max_sum)
      neuron_it->sum = max_sum;
    else if (neuron_it->sum < -max_sum)
      neuron_it->sum = -max_sum;

    fann_activation_switch(activation_function, neuron_it->sum, neuron_it->value);
    // printf("%d %.3f | ", threadIdx.x + work*work_per_thread, neuron_it->sum);

  }
}

void fann_backpropagate_MSE_custom(struct fann *ann, struct fann *dev_ann, fann_type *dev_weights, fann_type *dev_errors) {
  struct fann_layer *layer_it;
  struct fann_neuron *neuron_it, *last_neuron;
  struct fann_neuron **connections;
  unsigned int i;

  fann_type tmp_error;
  fann_type *error_begin = ann->train_errors;
  fann_type *error_prev_layer;
  fann_type *weights;
  const struct fann_neuron *first_neuron = ann->first_layer->first_neuron;
  const struct fann_layer *second_layer = ann->first_layer + 1;
  struct fann_layer *last_layer = ann->last_layer;

  /* go through all the layers, from last to first.
   * And propagate the error backwards */
  for (layer_it = last_layer - 1; layer_it > second_layer; --layer_it) {

    last_neuron = layer_it->last_neuron;
    int neurons_in_layer = layer_it->last_neuron - layer_it->first_neuron;
    int neurons_in_prev_layer = (layer_it-1)->last_neuron - (layer_it-1)->first_neuron;
    // printf("Layer with %d\n", neurons_in_layer);


    /* for each connection in this layer, propagate the error backwards */
    error_prev_layer = error_begin + ((layer_it - 1)->first_neuron - first_neuron);

    struct fann_neuron *dev_neurons, *dev_last_layer_neurons;
    check(hipMalloc((void **)&dev_neurons, neurons_in_layer*sizeof(struct fann_neuron)));
  	check(hipMemcpy(dev_neurons, layer_it->first_neuron, neurons_in_layer*sizeof(struct fann_neuron), hipMemcpyHostToDevice));

  	check(hipMalloc((void **)&dev_last_layer_neurons, neurons_in_prev_layer*sizeof(struct fann_neuron)));
  	check(hipMemcpy(dev_last_layer_neurons, (layer_it-1)->first_neuron, neurons_in_prev_layer*sizeof(struct fann_neuron), hipMemcpyHostToDevice));

    for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
      tmp_error = error_begin[neuron_it - first_neuron];
      weights = ann->weights + neuron_it->first_con;
      // printf("tmp_error: %f, n->it - first_neuron: %ld\n", tmp_error, neuron_it - first_neuron);
      for (i = neuron_it->last_con - neuron_it->first_con; i--;) {
        // printf("i: %d", i);
        // printf("w: %f\n", weights[i]);
        /*printf("i = %d\n", i);
         * printf("error_prev_layer[%d] = %f\n", i, error_prev_layer[i]);
         * printf("weights[%d] = %f\n", i, weights[i]); */
        error_prev_layer[i] += tmp_error * weights[i];
        // printf("%f\n", error_prev_layer[i]);
      }
    }
    // printf("\nCPU:\n");
    neuron_it = layer_it->first_neuron;
    for (i = neuron_it->last_con - neuron_it->first_con; i--;) {
      // printf("%f ", error_prev_layer[i]);
    }

    // for (neuron_it = layer_it->first_neuron; neuron_it != last_neuron; neuron_it++) {
    
    // }
    int first_neuron_number = 0, last_layer_first_neuron_number = 0;
  	struct fann_layer *l_it;
    for (l_it = layer_it-2; l_it >= ann->first_layer; l_it--) {
    	last_layer_first_neuron_number += l_it->last_neuron - l_it->first_neuron;
    }
    first_neuron_number = last_layer_first_neuron_number + (layer_it-1)->last_neuron - (layer_it-1)->first_neuron;

    error_begin = ann->train_errors + first_neuron_number;
    // printf("\nError %d\n", first_neuron_number);
    // for (int i = 0; i < 2; ++i)
    // {
    //   printf("%f | ", *(error_begin+i));  
    // }
    // printf("\nError ends\n");
    
    // printf("current:%d, prev:%d \n",first_neuron_number,last_layer_first_neuron_number);
    fann_backprop_layer<<<1, neurons_in_layer>>>(dev_weights, dev_neurons, first_neuron_number, 
      last_layer_first_neuron_number, dev_errors, neurons_in_layer);
    // hipDeviceSynchronize();
    // pr<<<1,1>>>(dev_errors, last_layer_first_neuron_number, neuron_it->last_con - neuron_it->first_con);
    // check(hipMemcpy(ann->train_errors, dev_errors, ));

    /* then calculate the actual errors in the previous layer */
    // error_prev_layer = error_begin + ((layer_it - 1)->first_neuron - first_neuron);
    // last_neuron = (layer_it - 1)->last_neuron;

    // printf("%p + %d = %p\n", dev_errors, ann->total_neurons*sizeof(fann_type), dev_errors+ann->total_neurons);
    // printf("%d\n", ann->total_neurons);
    fann_backprop_layer_complete<<<1, neurons_in_prev_layer>>>(dev_errors, dev_last_layer_neurons, 
      last_layer_first_neuron_number, neurons_in_prev_layer);
    // for (neuron_it = (layer_it - 1)->first_neuron; neuron_it != last_neuron; neuron_it++) {
    //   *error_prev_layer *=
    //       fann_activation_derived(neuron_it->activation_function, neuron_it->activation_steepness,
    //                               neuron_it->value, neuron_it->sum);
    //   error_prev_layer++;
    // }
    hipFree(dev_neurons);
    hipFree(dev_last_layer_neurons);
  }
}

__global__ void pr(fann_type *error_begin, int last_layer_first_neuron_number, int num_connections) {
  fann_type *error_prev_layer;
  unsigned int i;
  error_prev_layer = error_begin + last_layer_first_neuron_number;
  // weights = dev_weights + neuron_it->first_con;
  // printf("\ngpu:\n");
  for (i = num_connections; i--;) {
    // printf("%f ", error_prev_layer[i]);
  }
}

__global__ void fann_backprop_layer(fann_type *dev_weights, struct fann_neuron *dev_neurons, int first_neuron_number,
	int last_layer_first_neuron_number, fann_type *error_begin, int total_work) {
  // int work_per_thread = total_work/blockDim.x;
  // for (int work = 0; work < total_work; ++work)
  // {
    fann_type tmp_error;
    unsigned int i;
    fann_type *error_prev_layer, *weights;
  	struct fann_neuron *neuron_it = dev_neurons + threadIdx.x;
    // struct fann_neuron *neuron_it = dev_neurons + threadIdx.x*work_per_thread + work;
    tmp_error = error_begin[first_neuron_number + threadIdx.x];
    // tmp_error = error_begin[first_neuron_number + threadIdx.x*work_per_thread + work];
    // printf("tmp_error: %f, nv: %f\n", tmp_error, neuron_it->value);
    weights = dev_weights + neuron_it->first_con;
    error_prev_layer = error_begin + last_layer_first_neuron_number;
    // printf("%p\n", error_prev_layer);
    // printf("%u, %u\n", neuron_it->last_con, neuron_it->first_con);
    for (i = neuron_it->last_con - neuron_it->first_con; i--;) {
      atomicAdd(&error_prev_layer[i], tmp_error * weights[i]);
    }
  // }
}

__global__ void fann_backprop_layer_complete(fann_type *error_begin, struct fann_neuron *dev_last_layer_neurons, 
	int last_layer_first_neuron_number, int total_work) {
  // int work_per_thread = total_work/blockDim.x;
  // for (int work = 0; work < total_work; ++work)
  // {
  	struct fann_neuron *neuron_it = dev_last_layer_neurons + threadIdx.x;
    // struct fann_neuron *neuron_it = dev_last_layer_neurons + threadIdx.x*work_per_thread + work;
    fann_type *error_prev_layer;
    error_prev_layer = error_begin + last_layer_first_neuron_number + threadIdx.x;
    // error_prev_layer = error_begin + last_layer_first_neuron_number + threadIdx.x*work_per_thread + work;
    *error_prev_layer *= sigmoid_gradient(neuron_it->value, neuron_it->activation_steepness);
  // }
}

void fann_compute_MSE_custom(struct fann *ann, fann_type *desired_output, fann_type *desired_output_host, fann_type *dev_errors, struct fann *dev_ann) {
	fann_type *error_it = 0, *error_begin = 0;
	fann_type neuron_diff, neuron_value;
  struct fann_neuron *last_layer_begin = (ann->last_layer - 1)->first_neuron;
  // const struct fann_neuron *last_layer_end = last_layer_begin + ann->num_output;
  // const struct fann_neuron *first_neuron = ann->first_layer->first_neuron;

  int last_layer_first_neuron_number = 0;
	struct fann_layer *l_it;
  for (l_it = ann->last_layer-2; l_it >= ann->first_layer; l_it--) {
  	last_layer_first_neuron_number += l_it->last_neuron - l_it->first_neuron;
  }
	// const struct fann_neuron *last_layer_end = last_layer_begin + ann->num_output;
  // const struct fann_neuron *first_neuron = ann->first_layer->first_neuron;

  /* if no room allocated for the error variabels, allocate it now */
  // if (ann->train_errors == NULL) {
  //   ann->train_errors = (fann_type *)calloc(ann->total_neurons, sizeof(fann_type));
  //   if (ann->train_errors == NULL) {
  //     fann_error((struct fann_error *)ann, FANN_E_CANT_ALLOCATE_MEM);
  //     return;
  //   }
  // } 
  // else {
    /* clear the error variabels */
  check(hipMemset(dev_errors, 0, (ann->total_neurons) * sizeof(fann_type)));
  // memset(ann->train_errors, 0, (ann->total_neurons) * sizeof(fann_type));
	// }
  // error_begin = ann->train_errors;
	struct fann_neuron *dev_last_layer_begin;
  check(hipMalloc((void **)&dev_last_layer_begin, sizeof(struct fann_neuron)*ann->num_output));
	check(hipMemcpy(dev_last_layer_begin, last_layer_begin, sizeof(struct fann_neuron)*ann->num_output, hipMemcpyHostToDevice));
	// printf("\n");
#ifdef DEBUGTRAIN
  printf("\ncalculate errors\n");
#endif
  /* calculate the error and place it in the output layer */
  // error_it = error_begin + (last_layer_begin - first_neuron);

  // for (; last_layer_begin != last_layer_end; last_layer_begin++) {
  //   neuron_value = last_layer_begin->value;
  //   neuron_diff = *desired_output_host - neuron_value;

  //   neuron_diff = fann_update_MSE(ann, last_layer_begin, neuron_diff);

  //   if (ann->train_error_function) { /* TODO make switch when more functions */
  //     if (neuron_diff < -.9999999)
  //       neuron_diff = -17.0;
  //     else if (neuron_diff > .9999999)
  //       neuron_diff = 17.0;
  //     else
  //       neuron_diff = (fann_type)log((1.0 + neuron_diff) / (1.0 - neuron_diff));
  //   }

  //   *error_it = fann_activation_derived(last_layer_begin->activation_function,
  //                                       last_layer_begin->activation_steepness, neuron_value,
  //                                       last_layer_begin->sum) *
  //               neuron_diff;
  //   printf("cpu e:%f | n:%f | d:%f || ", *error_it, neuron_value, neuron_diff);
  //   desired_output_host++;
  //   error_it++;

  //   ann->num_MSE++;
  // }

  // printf("\n");
  calc_mse<<<1,ann->num_output>>>(dev_ann, dev_last_layer_begin, desired_output, dev_errors, last_layer_first_neuron_number, ann->num_output);



  check(hipMemcpy(&ann->MSE_value, &dev_ann->MSE_value, sizeof(float), hipMemcpyDeviceToHost));
  check(hipMemcpy(&ann->num_bit_fail, &dev_ann->num_bit_fail, sizeof(unsigned int), hipMemcpyDeviceToHost));

  check(hipMemcpy(ann->train_errors, dev_errors, (ann->total_neurons)*sizeof(fann_type), hipMemcpyDeviceToHost));
  error_begin = ann->train_errors + last_layer_first_neuron_number;
  // printf("\ncorrect Error %d \n", last_layer_first_neuron_number);
  // for (int i = 0; i < 2; ++i)
  // {
  //   printf("%f | ", *(error_begin+i));  
  // }
  // printf("\nError ends\n");
  ann->num_MSE += ann->num_output;

  hipFree(dev_last_layer_begin);
}

__global__ void calc_mse(struct fann *ann, struct fann_neuron *dev_last_layer_begin, fann_type *dev_output, 
	fann_type *error_begin, int last_layer_first_neuron_number, int total_work) {
  // int work_per_thread = total_work/blockDim.x;
  // for (int work = 0; work < total_work; ++work)
  // {
  	fann_type neuron_value, neuron_diff, *error_it;
  	fann_type *desired_output = dev_output + threadIdx.x;
    // fann_type *desired_output = dev_output + threadIdx.x*work_per_thread + work;
  	// printf("%f ", *desired_output, desired_output);
  	dev_last_layer_begin += threadIdx.x;
    // dev_last_layer_begin += threadIdx.x*work_per_thread + work;
  	
  	neuron_value = dev_last_layer_begin->value;
  	neuron_diff = *desired_output - neuron_value;
  	error_it = error_begin + last_layer_first_neuron_number + threadIdx.x;
    // error_it = error_begin + last_layer_first_neuron_number + threadIdx.x*work_per_thread + work;
  	
    // neuron_diff = fann_update_MSE(ann, last_layer_begin, neuron_diff);
    atomicAdd(&ann->MSE_value, neuron_diff*neuron_diff);
  	if (fann_abs(neuron_diff) >= ann->bit_fail_limit) {
  		atomicAdd(&ann->num_bit_fail, 1);
    }
  	
    if (ann->train_error_function) { /* TODO make switch when more functions */
      if (neuron_diff < -.9999999)
        neuron_diff = -17.0;
      else if (neuron_diff > .9999999)
        neuron_diff = 17.0;
      else
        neuron_diff = (fann_type)log((1.0 + neuron_diff) / (1.0 - neuron_diff));
    }
  	
    *error_it = sigmoid_gradient(neuron_value, dev_last_layer_begin->activation_steepness)*neuron_diff;
    // printf("kernel e:%f | n:%f | d:%f || ", *error_it, neuron_value, neuron_diff);
  // }
}

__device__ fann_type sigmoid_gradient(fann_type x, fann_type steepness) {
	const float lo = 0.01f;
	const float hi = 0.99f;
	x = (((x) < (lo)) ? (lo) : (((x) > (hi)) ? (hi) : (x)));
	return (2.0f * steepness * x * (1.0f - x));
}

void fann_update_weights_irpropm(struct fann *ann, unsigned int first_weight,
                                 unsigned int past_end) {
  fann_type *train_slopes = ann->train_slopes;
  fann_type *weights = ann->weights;
  fann_type *prev_steps = ann->prev_steps;
  fann_type *prev_train_slopes = ann->prev_train_slopes;
  fann_type prev_step, slope, prev_slope, next_step, same_sign;

  float increase_factor = ann->rprop_increase_factor; /*1.2; */
  float decrease_factor = ann->rprop_decrease_factor; /*0.5; */
  float delta_min = ann->rprop_delta_min;             /*0.0; */
  float delta_max = ann->rprop_delta_max;             /*50.0; */

  unsigned int i = first_weight;

  for (; i != past_end; i++) {
  	// printf("3\n");
  	// printf("%f\n", prev_steps[i]);
    prev_step = fann_max(
        prev_steps[i],
        (fann_type)0.0001); /* prev_step may not be zero because then the training will stop */
  	// printf("4\n");
    slope = train_slopes[i];
    prev_slope = prev_train_slopes[i];

    same_sign = prev_slope * slope;

    if (same_sign >= 0.0)
      next_step = fann_min(prev_step * increase_factor, delta_max);
    else {
      next_step = fann_max(prev_step * decrease_factor, delta_min);
      slope = 0;
    }

    if (slope < 0) {
      weights[i] -= next_step;
      if (weights[i] < -1500) weights[i] = -1500;
    } else {
      weights[i] += next_step;
      if (weights[i] > 1500) weights[i] = 1500;
    }

    /*if(i == 2){
     * printf("weight=%f, slope=%f, next_step=%f, prev_step=%f\n", weights[i], slope, next_step,
     * prev_step);
     * } */

    /* update global data arrays */
    prev_steps[i] = next_step;
    prev_train_slopes[i] = slope;
    train_slopes[i] = 0.0;
  }
}

void fann_update_slopes_batch_custom(struct fann *ann, struct fann_layer *layer_begin,
                              struct fann_layer *layer_end, fann_type *dev_slopes, fann_type *dev_errors) {
  // struct fann_neuron *last_neuron, *prev_neurons, **connections;
  // fann_type tmp_error;
  // unsigned int i, num_connections;

  /* store some variabels local for fast access */
  // struct fann_neuron *first_neuron = ann->first_layer->first_neuron;
  // fann_type *error_begin = ann->train_errors;
  // fann_type *slope_begin, *neuron_slope;

  /* if no room allocated for the slope variabels, allocate it now */

  if (layer_begin == NULL) {
    layer_begin = ann->first_layer + 1;
  }

  if (layer_end == NULL) {
    layer_end = ann->last_layer - 1;
  }

  // slope_begin = ann->train_slopes;

#ifdef DEBUGTRAIN
  printf("\nupdate slopes\n");
#endif

  // prev_neurons = first_neuron;

  for (; layer_begin <= layer_end; layer_begin++) {
#ifdef DEBUGTRAIN
    printf("layer[%d]\n", layer_begin - ann->first_layer);
#endif
    // last_neuron = layer_begin->last_neuron;

    int neurons_in_layer = layer_begin->last_neuron - layer_begin->first_neuron;
    int neurons_in_prev_layer = (layer_begin-1)->last_neuron - (layer_begin-1)->first_neuron;

    // if (ann->network_type == FANN_NETTYPE_LAYER) {
    // prev_neurons = (layer_begin - 1)->first_neuron;
    // }
    struct fann_neuron *dev_neurons, *dev_last_layer_neurons;
    check(hipMalloc((void **)&dev_neurons, neurons_in_layer*sizeof(struct fann_neuron)));
  	check(hipMemcpy(dev_neurons, layer_begin->first_neuron, neurons_in_layer*sizeof(struct fann_neuron), hipMemcpyHostToDevice));

  	check(hipMalloc((void **)&dev_last_layer_neurons, neurons_in_prev_layer*sizeof(struct fann_neuron)));
  	check(hipMemcpy(dev_last_layer_neurons, (layer_begin-1)->first_neuron, neurons_in_prev_layer*sizeof(struct fann_neuron), hipMemcpyHostToDevice));

  	// position of the first neuron in this layer == total neurons upto this layer
  	int first_neuron_number = 0;
		struct fann_layer *l_it;
	  for (l_it = layer_begin-1; l_it >= ann->first_layer; l_it--) {
	  	first_neuron_number += l_it->last_neuron - l_it->first_neuron;
	  }
    // printf("%d\n", neurons_in_layer);
	  update_slopes<<<1, (int)(neurons_in_layer/scale_threads)>>>(dev_neurons, dev_last_layer_neurons, first_neuron_number, dev_slopes, dev_errors, neurons_in_layer);

    // for (neuron_it = layer_begin->first_neuron; neuron_it != last_neuron; neuron_it++) {
    // }
    hipFree(dev_neurons);
    hipFree(dev_last_layer_neurons);
  }
}

__global__ void update_slopes(struct fann_neuron *dev_neurons, struct fann_neuron *dev_last_layer_neurons, int first_neuron_number, 
	fann_type *dev_slopes, fann_type *dev_errors, int total_work) {
  // int work_per_thread = total_work/blockDim.x;
  // for (int work = 0; work < total_work; ++work)
  // {
  	struct fann_neuron *neuron_it;
  	fann_type *neuron_slope, tmp_error;
  	unsigned int i;
  	int num_connections;
  	neuron_it = dev_neurons + threadIdx.x;
    // neuron_it = dev_neurons + threadIdx.x + work_per_thread*work;
  	tmp_error = dev_errors[first_neuron_number + threadIdx.x];
    // tmp_error = dev_errors[first_neuron_number + threadIdx.x*work_per_thread + work];
    neuron_slope = dev_slopes + neuron_it->first_con;
    num_connections = neuron_it->last_con - neuron_it->first_con;
    for (i = 0; i != num_connections; i++) {
      atomicAdd(&neuron_slope[i], tmp_error * dev_last_layer_neurons[i].value);
    }
  // }
}


fann_type fann_update_MSE(struct fann *ann, struct fann_neuron *neuron, fann_type neuron_diff) {
  float neuron_diff2;

  switch (neuron->activation_function) {
    case FANN_LINEAR_PIECE_SYMMETRIC:
    case FANN_THRESHOLD_SYMMETRIC:
    case FANN_SIGMOID_SYMMETRIC:
    case FANN_SIGMOID_SYMMETRIC_STEPWISE:
    case FANN_ELLIOT_SYMMETRIC:
    case FANN_GAUSSIAN_SYMMETRIC:
    case FANN_SIN_SYMMETRIC:
    case FANN_COS_SYMMETRIC:
      neuron_diff /= (fann_type)2.0;
      break;
    case FANN_THRESHOLD:
    case FANN_LINEAR:
    case FANN_SIGMOID:
    case FANN_SIGMOID_STEPWISE:
    case FANN_GAUSSIAN:
    case FANN_GAUSSIAN_STEPWISE:
    case FANN_ELLIOT:
    case FANN_LINEAR_PIECE:
    case FANN_SIN:
    case FANN_COS:
      break;
  }

#ifdef FIXEDFANN
  neuron_diff2 = (neuron_diff / (float)ann->multiplier) * (neuron_diff / (float)ann->multiplier);
#else
  neuron_diff2 = (float)(neuron_diff * neuron_diff);
#endif

  ann->MSE_value += neuron_diff2;

  /*printf("neuron_diff %f = (%f - %f)[/2], neuron_diff2=%f, sum=%f, MSE_value=%f, num_MSE=%d\n",
   * neuron_diff, *desired_output, neuron_value, neuron_diff2, last_layer_begin->sum,
   * ann->MSE_value, ann->num_MSE); */
  if (fann_abs(neuron_diff) >= ann->bit_fail_limit) {
    ann->num_bit_fail++;
  }

  return neuron_diff;
}